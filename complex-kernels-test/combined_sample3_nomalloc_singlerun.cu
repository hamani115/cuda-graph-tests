#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <cmath>    // For sqrt in standard deviation calculation
#include <algorithm> // For std::find
#include <chrono>


#include <fstream>
#include <string>
#include <sstream>

// Local headers
#include "../cuda_check.h"
// #include "../csv_util.h"

#define DEFAULT_NSTEP 100000
#define DEFAULT_SKIPBY 0

// Kernel functions
__global__ void kernelA(double* arrayA, size_t size){
    const size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    if(x < size){ arrayA[x] *= 2.0; }
}

__global__ void kernelB(int* arrayB, size_t size){
    const size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    if(x < size){ arrayB[x] = 3; }
}

__global__ void kernelC(double* arrayA, const int* arrayB, size_t size){
    const size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    if(x < size){ arrayA[x] += arrayB[x]; }
}

struct CSVData {
    int NSTEP;
    int SKIPBY;
    float noneGraphTotalTimeWithout;
    float GraphTotalTimeWithout;
    float noneGraphTotalTimeWith;
    float GraphTotalTimeWith;
    float DiffTotalWithout;
    float DiffPerStepWithout;
    float DiffPercentWithout;
    float DiffTotalWith;
    float DiffPerStepWith;
    float DiffPercentWith;
    float ChronoNoneGraphTotalTimeWithout;
    float ChronoGraphTotalTimeWithout;
    float ChronoNoneGraphTotalLaunchTimeWithout;
    float ChronoGraphTotalLaunchTimeWithout;
    float ChronoNoneGraphTotalTimeWith;
    float ChronoGraphTotalTimeWith;
    float ChronoNoneGraphTotalLaunchTimeWith;
    float ChronoGraphTotalLaunchTimeWith;
    float ChronoDiffTotalTimeWithout;
    float ChronoDiffPerStepWithout;
    float ChronoDiffPercentWithout;
    float ChronoDiffTotalTimeWith;
    float ChronoDiffPerStepWith;
    float ChronoDiffPercentWith;
    float ChronoDiffLaunchTimeWithout;
    float ChronoDiffLaunchPercentWithout;
    float ChronoDiffLaunchTimeWith;
    float ChronoDiffLaunchPercentWith;
};

// Function to update or append data in CSV
void updateOrAppendCSV(const std::string &filename, const CSVData &newData) {
    std::vector<CSVData> csvData;
    std::ifstream csvFileIn(filename);
    if (csvFileIn.is_open()) {
        std::string line;
        // Skip the header line
        std::getline(csvFileIn, line);
        while (std::getline(csvFileIn, line)) {
            std::istringstream ss(line);
            CSVData data;
            std::string token;

            // Parse each field in order
            if (std::getline(ss, token, ',')) data.NSTEP = std::stoi(token);
            if (std::getline(ss, token, ',')) data.SKIPBY = std::stoi(token);
            if (std::getline(ss, token, ',')) data.noneGraphTotalTimeWithout = std::stof(token);
            if (std::getline(ss, token, ',')) data.GraphTotalTimeWithout = std::stof(token);
            if (std::getline(ss, token, ',')) data.noneGraphTotalTimeWith = std::stof(token);
            if (std::getline(ss, token, ',')) data.GraphTotalTimeWith = std::stof(token);
            if (std::getline(ss, token, ',')) data.DiffTotalWithout = std::stof(token);
            if (std::getline(ss, token, ',')) data.DiffPerStepWithout = std::stof(token);
            if (std::getline(ss, token, ',')) data.DiffPercentWithout = std::stof(token);
            if (std::getline(ss, token, ',')) data.DiffTotalWith = std::stof(token);
            if (std::getline(ss, token, ',')) data.DiffPerStepWith = std::stof(token);
            if (std::getline(ss, token, ',')) data.DiffPercentWith = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoNoneGraphTotalTimeWithout = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoGraphTotalTimeWithout = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoNoneGraphTotalLaunchTimeWithout = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoGraphTotalLaunchTimeWithout = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoNoneGraphTotalTimeWith = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoGraphTotalTimeWith = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoNoneGraphTotalLaunchTimeWith = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoGraphTotalLaunchTimeWith = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoDiffTotalTimeWithout = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoDiffPerStepWithout = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoDiffPercentWithout = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoDiffTotalTimeWith = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoDiffPerStepWith = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoDiffPercentWith = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoDiffLaunchTimeWithout = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoDiffLaunchPercentWithout = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoDiffLaunchTimeWith = std::stof(token);
            if (std::getline(ss, token, ',')) data.ChronoDiffLaunchPercentWith = std::stof(token);

            csvData.push_back(data);
        }
        csvFileIn.close();
    }

    // Update or append
    bool updated = false;
    for (auto &entry : csvData) {
        if (entry.NSTEP == newData.NSTEP && entry.SKIPBY == newData.SKIPBY) {
            entry = newData;
            updated = true;
            break;
        }
    }

    if (!updated) {
        csvData.push_back(newData);
    }

    std::string tempFILENAME = "complex_3_different_kernels.tmp";
    {
        std::ofstream tempFile(tempFILENAME);
        if (!tempFile.is_open()) {
            std::cerr << "Failed to open the temporary file for writing!" << std::endl;
            return;
        }

        tempFile << "NSTEP,SKIPBY,"
                    "noneGraphTotalTimeWithout,GraphTotalTimeWithout,"
                    "noneGraphTotalTimeWith,GraphTotalTimeWith,"
                    "DiffTotalWithout,DiffPerStepWithout,DiffPercentWithout,"
                    "DiffTotalWith,DiffPerStepWith,DiffPercentWith,"
                    "ChronoNoneGraphTotalTimeWithout,ChronoGraphTotalTimeWithout,"
                    "ChronoNoneGraphTotalLaunchTimeWithout,ChronoGraphTotalLaunchTimeWithout,"
                    "ChronoNoneGraphTotalTimeWith,ChronoGraphTotalTimeWith,"
                    "ChronoNoneGraphTotalLaunchTimeWith,ChronoGraphTotalLaunchTimeWith,"
                    "ChronoDiffTotalTimeWithout,ChronoDiffPerStepWithout,ChronoDiffPercentWithout,"
                    "ChronoDiffTotalTimeWith,ChronoDiffPerStepWith,ChronoDiffPercentWith,"
                    "ChronoDiffLaunchTimeWithout,ChronoDiffLaunchPercentWithout,"
                    "ChronoDiffLaunchTimeWith,ChronoDiffLaunchPercentWith\n";

        for (const auto &entry : csvData) {
            tempFile << entry.NSTEP << ","
                     << entry.SKIPBY << ","
                     << entry.noneGraphTotalTimeWithout << ","
                     << entry.GraphTotalTimeWithout << ","
                     << entry.fanoneGraphTotalTimeWith << ","
                     << entry.GraphTotalTimeWith << ","
                     << entry.DiffTotalWithout << ","
                     << entry.DiffPerStepWithout << ","
                     << entry.DiffPercentWithout << ","
                     << entry.DiffTotalWith << ","
                     << entry.DiffPerStepWith << ","
                     << entry.DiffPercentWith << ","
                     << entry.ChronoNoneGraphTotalTimeWithout << ","
                     << entry.ChronoGraphTotalTimeWithout << ","
                     << entry.ChronoNoneGraphTotalLaunchTimeWithout << ","
                     << entry.ChronoGraphTotalLaunchTimeWithout << ","
                     << entry.ChronoNoneGraphTotalTimeWith << ","
                     << entry.ChronoGraphTotalTimeWith << ","
                     << entry.ChronoNoneGraphTotalLaunchTimeWith << ","
                     << entry.ChronoGraphTotalLaunchTimeWith << ","
                     << entry.ChronoDiffTotalTimeWithout << ","
                     << entry.ChronoDiffPerStepWithout << ","
                     << entry.ChronoDiffPercentWithout << ","
                     << entry.ChronoDiffTotalTimeWith << ","
                     << entry.ChronoDiffPerStepWith << ","
                     << entry.ChronoDiffPercentWith << ","
                     << entry.ChronoDiffLaunchTimeWithout << ","
                     << entry.ChronoDiffLaunchPercentWithout << ","
                     << entry.ChronoDiffLaunchTimeWith << ","
                     << entry.ChronoDiffLaunchPercentWith
                     << "\n";
        }
    }

    std::remove(filename.c_str());
    std::rename(tempFILENAME.c_str(), filename.c_str());
    std::cout << "SUCCESS: ADDED/UPDATED CSV FILE\n";
}



// struct set_vector_args {
//     double* h_array;
//     double value;
//     size_t size;
// };

// void CUDART_CB set_vector(void* args) {
//     set_vector_args* h_args = reinterpret_cast<set_vector_args*>(args);
//     double* array = h_args->h_array;
//     size_t size = h_args->size;
//     double value = h_args->value;

//     // Initialize h_array with the specified value
//     for (size_t i = 0; i < size; ++i) {
//         array[i] = value;
//     }

//     // Do NOT delete h_args here
// }

std::vector<int> generateSequence(int N) {
    std::vector<int> sequence;
    int current = 5; // Starting point
    bool multiplyByTwo = true; // Flag to alternate between multiplying by 2 and 5
    
    while (current <= N) {
        sequence.push_back(current);
        
        if (multiplyByTwo) {
            current *= 2;
        } else {
            current *= 5;
        }
        
        multiplyByTwo = !multiplyByTwo; // Toggle the multiplier for next iteration
    }
    
    return sequence;
}

// Function for non-graph implementation
// void runWithoutGraph(float* totalTimeWith, float* totalTimeWithout, int nstep, int skipby) {
void runWithoutGraph(std::vector<float>& totalTimeWithArr, std::vector<float>& totalTimeWithoutArr, std::vector<float>& chronoTotalTimeWithArr, std::vector<float>& chronoTotalTimeWithoutArr, std::vector<float>& chronoTotalLaunchTimeWithArr,std::vector<float>& chronoTotalLaunchTimeWithoutArr,
                int nstep, int skipby) {
    // Declare constants inside the function
    const int NSTEP = nstep;
    const int SKIPBY = skipby;

    constexpr int numOfBlocks = 1024;
    constexpr int threadsPerBlock = 1024;
    constexpr size_t arraySize = 1U << 20;
    // constexpr int iterations = 1000;
    constexpr double initValue = 2.0;

    // Host and device memory
    double* d_arrayA;
    int* d_arrayB;
    // std::vector<double> h_array(arraySize);
    double* h_array = nullptr;
    CUDA_CHECK(hipHostMalloc((void**)&h_array, arraySize * sizeof(double)));

    // Initialize host array using index i
    for (size_t i = 0; i < arraySize; i++) {
        h_array[i] = initValue;
    }

    // Set Timer for first run
    hipEvent_t firstCreateStart, firstCreateStop;
    float firstCreateTime = 0.0f;
    CUDA_CHECK(hipEventCreate(&firstCreateStart));
    CUDA_CHECK(hipEventCreate(&firstCreateStop));

    // Create a stream
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Allocate device memory
    CUDA_CHECK(hipMallocAsync(&d_arrayA, arraySize * sizeof(double), stream));
    CUDA_CHECK(hipMallocAsync(&d_arrayB, arraySize * sizeof(int), stream));

    // Start measuring first run time
    CUDA_CHECK(hipEventRecord(firstCreateStart, stream));
    const auto graphStart = std::chrono::steady_clock::now();

    // Copy h_array to device
    CUDA_CHECK(hipMemcpyAsync(d_arrayA, h_array, arraySize * sizeof(double), hipMemcpyHostToDevice, stream));

    // Launch kernels
    kernelA<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayA, arraySize);
    kernelB<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayB, arraySize);
    kernelC<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayA, d_arrayB, arraySize);

    // Copy data back to host
    CUDA_CHECK(hipMemcpyAsync(h_array, d_arrayA, arraySize * sizeof(double), hipMemcpyDeviceToHost, stream));

    // Wait for all operations to complete
    // CUDA_CHECK(hipStreamSynchronize(stream));

    const auto graphEnd = std::chrono::steady_clock::now();
    // Stop measuring first run time
    CUDA_CHECK(hipEventRecord(firstCreateStop, stream));
    CUDA_CHECK(hipEventSynchronize(firstCreateStop));
    const auto graphEnd2 = std::chrono::steady_clock::now();
    CUDA_CHECK(hipEventElapsedTime(&firstCreateTime, firstCreateStart, firstCreateStop));
    const std::chrono::duration<double> graphCreateTimeChrono = graphEnd - graphStart;
    const std::chrono::duration<double> graphCreateTimeChrono2 = graphEnd2 - graphStart;

    // Measure execution time
    hipEvent_t execStart, execStop;
    CUDA_CHECK(hipEventCreate(&execStart));
    CUDA_CHECK(hipEventCreate(&execStop));

    float elapsedTime = 0.0f;
    float totalTime = 0.0f;
    std::chrono::duration<double> totalTimeChrono = std::chrono::duration<double>(0.0);
    std::chrono::duration<double> totalLunchTimeChrono = std::chrono::duration<double>(0.0);
    float upperTime = 0.0f;
    float lowerTime = 0.0f;
    // int skipBy = 0;
    double mean = 0.0;
    double M2 = 0.0;
    int count = 0;

    std::vector<int> nsteps = generateSequence(NSTEP);

    // Execute the sequence multiple times
    for(int i = 1; i <= NSTEP; i++){

        for (size_t j = 0; j < arraySize; j++) {
            h_array[j] = initValue;
        }

        CUDA_CHECK(hipEventRecord(execStart, stream));
        const auto start = std::chrono::steady_clock::now();

        // Initialize host array using index i
        // for (size_t j = 0; j < arraySize; ++j) {
        //     h_array[j] = static_cast<double>(j);
        // }
        // Initialize host array
        // h_array.assign(h_array.size(), initValue);

        // Copy h_array to device
        CUDA_CHECK(hipMemcpyAsync(d_arrayA, h_array, arraySize * sizeof(double), hipMemcpyHostToDevice, stream));

        // Launch kernels
        kernelA<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayA, arraySize);
        kernelB<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayB, arraySize);
        kernelC<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayA, d_arrayB, arraySize);

        // Copy data back to host
        CUDA_CHECK(hipMemcpyAsync(h_array, d_arrayA, arraySize * sizeof(double), hipMemcpyDeviceToHost, stream));

        // Wait for all operations to complete
        // CUDA_CHECK(hipStreamSynchronize(stream));

        const auto end = std::chrono::steady_clock::now();
        CUDA_CHECK(hipEventRecord(execStop, stream));
        CUDA_CHECK(hipEventSynchronize(execStop));
        const auto end2 = std::chrono::steady_clock::now();
        CUDA_CHECK(hipEventElapsedTime(&elapsedTime, execStart, execStop));

        // Time calculations
        if (i >= SKIPBY) {
            const std::chrono::duration<double> launchTimeChrono = end - start;
            const std::chrono::duration<double> lunchExecTimeChrono = end2 - start;
            totalTimeChrono += lunchExecTimeChrono;
            totalLunchTimeChrono += launchTimeChrono;
            totalTime += elapsedTime;

            // Welford's algorithm for calculating mean and variance
            count++;
            double delta = elapsedTime - mean;
            mean += delta / count;
            double delta2 = elapsedTime - mean;
            M2 += delta * delta2;

            if (elapsedTime > upperTime) {
                upperTime = elapsedTime;
            }
            if (elapsedTime < lowerTime || lowerTime == 0.0f) {
                lowerTime = elapsedTime;
            }

            for (const auto& num : nsteps) {
                if (num == i) {
                    // Calculate mean and standard deviation
                    float meanTime = (totalTime + firstCreateTime) / (i + 1 - SKIPBY);
                    double varianceTime = (count > 1) ? M2 / (count - 1) : 0.0;
                    double stdDevTime = sqrt(varianceTime);

                    // Print out the time statistics
                    std::cout << "=======Setup (No Graph) for NSTEP " << i << "=======" << std::endl;
                    std::cout << "Iterations: " << NSTEP << std::endl;
                    std::cout << "Skip By: " << SKIPBY << std::endl;
                    std::cout << "Kernels: kernelA, kernelB, kernelC" << std::endl;
                    std::cout << "Number of Blocks: " << numOfBlocks << std::endl;
                    std::cout << "Threads per Block: " << threadsPerBlock << std::endl;
                    std::cout << "Array Size: " << arraySize << std::endl;
                    std::cout << "=======Results (No Graph) for NSTEP " << i << "=======" << std::endl;
                    std::cout << "First Run: " << firstCreateTime << " ms" << std::endl;
                    std::cout << "Average Time with firstRun: " << meanTime << " ms" << std::endl;
                    std::cout << "Average Time without firstRun: " << (totalTime / (i - SKIPBY)) << " ms" << std::endl;
                    std::cout << "Variance: " << varianceTime << " ms^2" << std::endl;
                    std::cout << "Standard Deviation: " << stdDevTime << " ms" << std::endl;
                    std::cout << "Time Spread: " << lowerTime << " - " << upperTime << " ms" << std::endl;
                    std::cout << "Total Time without firstRun: " << totalTime << " ms" << std::endl;
                    std::cout << "Total Time with firstRun: " << totalTime + firstCreateTime << " ms" << std::endl;
                    // CHRONO
                    const std::chrono::duration<double> totalTimeWithChrono = totalTimeChrono + graphCreateTimeChrono2;
                    const std::chrono::duration<double> totalLunchTimeWithChrono = totalLunchTimeChrono + graphCreateTimeChrono;
                    std::cout << "Graph Creation Chrono Launch before Sync: " << graphCreateTimeChrono.count() * 1000  << " ms" << std::endl;
                    std::cout << "Graph Creation Chrono LunchExec after Sync: " << graphCreateTimeChrono2.count() * 1000  << " ms" << std::endl;
                    std::cout << "Total Time Chrono Launch without Graph Creation: " << totalLunchTimeChrono.count() * 1000  << " ms" << std::endl;
                    std::cout << "Total Time Chrono Launch with Graph Creation: " << totalLunchTimeWithChrono.count() * 1000  << " ms" << std::endl;
                    std::cout << "Total Time Chrono LunchExec without Graph Creation: " << totalTimeChrono.count() * 1000  << " ms" << std::endl;
                    std::cout << "Total Time Chrono LunchExec with Graph Creation: " << totalTimeWithChrono.count() * 1000 << " ms" << std::endl;
                    
                    totalTimeWithArr.push_back(totalTime + firstCreateTime);
                    totalTimeWithoutArr.push_back(totalTime);
                    chronoTotalTimeWithArr.push_back(totalTimeWithChrono.count() * 1000);
                    chronoTotalTimeWithoutArr.push_back(totalTimeChrono.count() * 1000);
                    chronoTotalLaunchTimeWithArr.push_back(totalLunchTimeWithChrono.count() * 1000);
                    chronoTotalLaunchTimeWithoutArr.push_back(totalLunchTimeChrono.count() * 1000);
                }
            }
        }
    }

    // // Calculate mean and standard deviation
    // float meanTime = (totalTime + firstCreateTime) / NSTEP;
    // double varianceTime = (count > 1) ? M2 / (count - 1) : 0.0;
    // double stdDevTime = sqrt(varianceTime);

    // // Print out the time statistics
    // std::cout << "=======Setup (No Graph)=======" << std::endl;
    // std::cout << "Iterations: " << NSTEP << std::endl;
    // std::cout << "Skip By: " << SKIPBY << std::endl;
    // std::cout << "Kernels: kernelA, kernelB, kernelC" << std::endl;
    // std::cout << "Number of Blocks: " << numOfBlocks << std::endl;
    // std::cout << "Threads per Block: " << threadsPerBlock << std::endl;
    // std::cout << "Array Size: " << arraySize << std::endl;
    // std::cout << "=======Results (No Graph)=======" << std::endl;
    // std::cout << "First Run: " << firstCreateTime << " ms" << std::endl;
    // std::cout << "Average Time with firstRun: " << meanTime << " ms" << std::endl;
    // std::cout << "Average Time without firstRun: " << (totalTime / (NSTEP - 1 - SKIPBY)) << " ms" << std::endl;
    // std::cout << "Variance: " << varianceTime << " ms^2" << std::endl;
    // std::cout << "Standard Deviation: " << stdDevTime << " ms" << std::endl;
    // std::cout << "Time Spread: " << lowerTime << " - " << upperTime << " ms" << std::endl;
    // std::cout << "Total Time without firstRun: " << totalTime << " ms" << std::endl;
    // std::cout << "Total Time with firstRun: " << totalTime + firstCreateTime << " ms" << std::endl;

    // Verify results
    constexpr double expected = initValue * 2.0 + 3;
    bool passed = true;
    for(size_t i = 0; i < arraySize; i++){
        if(h_array[i] != expected){
            passed = false;
            std::cerr << "Validation failed! Expected " << expected << " got " << h_array[i] << " at index " << i << std::endl;
            break;
        }
    }
    if(passed){
        std::cerr << "Validation passed." << std::endl;
    }

    // Free device memory
    CUDA_CHECK(hipFreeAsync(d_arrayA, stream));
    CUDA_CHECK(hipFreeAsync(d_arrayB, stream));
    // Clean up
    CUDA_CHECK(hipEventDestroy(execStart));
    CUDA_CHECK(hipEventDestroy(execStop));
    CUDA_CHECK(hipEventDestroy(firstCreateStart));
    CUDA_CHECK(hipEventDestroy(firstCreateStop));
    CUDA_CHECK(hipStreamDestroy(stream));

    // Return total time including first run
    // return totalTime + firstCreateTime;
    // *totalTimeWith = totalTime + firstCreateTime;
    // *totalTimeWithout = totalTime;
}

// Function for graph implementation
// void runWithGraph(float* totalTimeWith, float* totalTimeWithout, int nstep, int skipby) {
void runWithGraph(std::vector<float>& totalTimeWithArr, std::vector<float>& totalTimeWithoutArr, std::vector<float>& chronoTotalTimeWithArr, std::vector<float>& chronoTotalTimeWithoutArr, std::vector<float>& chronoTotalLaunchTimeWithArr, std::vector<float>& chronoTotalLaunchTimeWithoutArr,
                int nstep, int skipby) {
    // Declare constants inside the function
    const int NSTEP = nstep;
    const int SKIPBY = skipby;
    
    constexpr int numOfBlocks = 1024;
    constexpr int threadsPerBlock = 1024;
    constexpr size_t arraySize = 1U << 20;
    // constexpr int iterations = 1000;
    constexpr double initValue = 2.0;

    double* d_arrayA;
    int* d_arrayB;
    double* h_array = nullptr;
    CUDA_CHECK(hipHostMalloc((void**)&h_array, arraySize * sizeof(double)));

    // Initialize host array using index i
    for (size_t i = 0; i < arraySize; i++) {
        h_array[i] = initValue;
    }

    // Set Timer for graph creation
    hipEvent_t graphCreateStart, graphCreateStop;
    float graphCreateTime = 0.0f;
    CUDA_CHECK(hipEventCreate(&graphCreateStart));
    CUDA_CHECK(hipEventCreate(&graphCreateStop));

    hipStream_t captureStream;
    CUDA_CHECK(hipStreamCreate(&captureStream));

    // Allocate device memory asynchronously
    CUDA_CHECK(hipMallocAsync(&d_arrayA, arraySize * sizeof(double), captureStream));
    CUDA_CHECK(hipMallocAsync(&d_arrayB, arraySize * sizeof(int), captureStream));

    // Start measuring graph creation time
    CUDA_CHECK(hipEventRecord(graphCreateStart, captureStream));
    const auto graphStart = std::chrono::steady_clock::now();

    // Start capturing operations
    CUDA_CHECK(hipStreamBeginCapture(captureStream, hipStreamCaptureModeGlobal));

    // set_vector_args* args = new set_vector_args{h_array, initValue, arraySize};
    // CUDA_CHECK(hipLaunchHostFunc(captureStream, set_vector, args));

    // Copy h_array to device
    CUDA_CHECK(hipMemcpyAsync(d_arrayA, h_array, arraySize * sizeof(double), hipMemcpyHostToDevice, captureStream));

    // Launch kernels
    kernelA<<<numOfBlocks, threadsPerBlock, 0, captureStream>>>(d_arrayA, arraySize);
    kernelB<<<numOfBlocks, threadsPerBlock, 0, captureStream>>>(d_arrayB, arraySize);
    kernelC<<<numOfBlocks, threadsPerBlock, 0, captureStream>>>(d_arrayA, d_arrayB, arraySize);

    // Copy data back to host
    CUDA_CHECK(hipMemcpyAsync(h_array, d_arrayA, arraySize * sizeof(double), hipMemcpyDeviceToHost, captureStream));

    // Stop capturing
    hipGraph_t graph;
    CUDA_CHECK(hipStreamEndCapture(captureStream, &graph));

    // Create an executable graph
    hipGraphExec_t graphExec;
    CUDA_CHECK(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

    // Destroy the graph template if not needed
    CUDA_CHECK(hipGraphDestroy(graph));

    // First Graph Launch

    const auto graphEnd = std::chrono::steady_clock::now();
    // Stop measuring graph creation time
    CUDA_CHECK(hipEventRecord(graphCreateStop, captureStream));
    CUDA_CHECK(hipEventSynchronize(graphCreateStop));
    const auto graphEnd2 = std::chrono::steady_clock::now();
    CUDA_CHECK(hipEventElapsedTime(&graphCreateTime, graphCreateStart, graphCreateStop));
    const std::chrono::duration<double> graphCreateTimeChrono = graphEnd - graphStart;
    const std::chrono::duration<double> graphCreateTimeChrono2 = graphEnd2 - graphStart;

    // Measure execution time
    hipEvent_t execStart, execStop;
    CUDA_CHECK(hipEventCreate(&execStart));
    CUDA_CHECK(hipEventCreate(&execStop));

    float elapsedTime = 0.0f;
    float totalTime = 0.0f;
    std::chrono::duration<double> totalTimeChrono = std::chrono::duration<double>(0.0);
    std::chrono::duration<double> totalLunchTimeChrono = std::chrono::duration<double>(0.0);
    float upperTime = 0.0f;
    float lowerTime = 0.0f;
    // int skipBy = 0;
    double mean = 0.0;
    double M2 = 0.0;
    int count = 0;

    // std::vector<int> nsteps_base = {10000, 5000, 1000, 500, 100, 50, 10, 5};
    std::vector<int> nsteps = generateSequence(NSTEP);
    
    // Filter NSTEP values based on input
    // for (int ns : nsteps_base) {
    //     if (ns <= NSTEP) {
    //         nsteps.push_back(ns);
    //     }
    // }

    // Launch the graph multiple times
    for(int i = 1; i <= NSTEP; i++){

        for (size_t j = 0; j < arraySize; j++) {
            h_array[j] = initValue;
        }
        
        CUDA_CHECK(hipEventRecord(execStart, captureStream));
        const auto start = std::chrono::steady_clock::now();

        CUDA_CHECK(hipGraphLaunch(graphExec, captureStream));
        // CUDA_CHECK(hipStreamSynchronize(captureStream));

        const auto end = std::chrono::steady_clock::now();
        CUDA_CHECK(hipEventRecord(execStop, captureStream));
        CUDA_CHECK(hipEventSynchronize(execStop));
        const auto end2 = std::chrono::steady_clock::now();
        CUDA_CHECK(hipEventElapsedTime(&elapsedTime, execStart, execStop));

        // Time calculations
        if (i >= SKIPBY) {
            const std::chrono::duration<double> launchTimeChrono = end - start;
            const std::chrono::duration<double> lunchExecTimeChrono = end2 - start;
            totalTimeChrono += lunchExecTimeChrono;
            totalLunchTimeChrono += launchTimeChrono;
            totalTime += elapsedTime;

            // Welford's algorithm for calculating mean and variance
            count++;
            double delta = elapsedTime - mean;
            mean += delta / count;
            double delta2 = elapsedTime - mean;
            M2 += delta * delta2;

            if (elapsedTime > upperTime) {
                upperTime = elapsedTime;
            }
            if (elapsedTime < lowerTime || lowerTime == 0.0f) {
                lowerTime = elapsedTime;
            }

            for (const auto& num : nsteps) {
                if (num == i) {
                    // Calculate mean and standard deviation
                    float meanTime = (totalTime + graphCreateTime) / (i + 1 - SKIPBY);
                    double varianceTime = (count > 1) ? M2 / (count - 1) : 0.0;
                    double stdDevTime = sqrt(varianceTime);

                    // Print out the time statistics
                    std::cout << "=======Setup (With Graph) for NSTEP " << i << "=======" << std::endl;
                    std::cout << "Iterations: " << i << std::endl;
                    std::cout << "Skip By: " << SKIPBY << std::endl;
                    std::cout << "Kernels: kernelA, kernelB, kernelC" << std::endl;
                    std::cout << "Number of Blocks: " << numOfBlocks << std::endl;
                    std::cout << "Threads per Block: " << threadsPerBlock << std::endl;
                    std::cout << "Array Size: " << arraySize << std::endl;
                    std::cout << "=======Results (With Graph) for NSTEP " << i << "=======" << std::endl;
                    std::cout << "Graph Creation Time: " << graphCreateTime << " ms" << std::endl;
                    std::cout << "Average Time with Graph: " << meanTime << " ms" << std::endl;
                    std::cout << "Average Time without Graph: " << (totalTime / (i - SKIPBY)) << " ms" << std::endl;
                    std::cout << "Variance: " << varianceTime << " ms^2" << std::endl;
                    std::cout << "Standard Deviation: " << stdDevTime << " ms" << std::endl;
                    std::cout << "Time Spread: " << lowerTime << " - " << upperTime << " ms" << std::endl;
                    std::cout << "Total Time without Graph Creation: " << totalTime << " ms" << std::endl;
                    std::cout << "Total Time with Graph Creation: " << totalTime + graphCreateTime << " ms" << std::endl;
                    // CHRONO
                    const std::chrono::duration<double> totalTimeWithChrono = totalTimeChrono + graphCreateTimeChrono2;
                    const std::chrono::duration<double> totalLunchTimeWithChrono = totalLunchTimeChrono + graphCreateTimeChrono;
                    std::cout << "Graph Creation Chrono Launch before Sync: " << graphCreateTimeChrono.count() * 1000  << " ms" << std::endl;
                    std::cout << "Graph Creation Chrono LunchExec after Sync: " << graphCreateTimeChrono2.count() * 1000  << " ms" << std::endl;
                    std::cout << "Total Time Chrono Launch without Graph Creation: " << totalLunchTimeChrono.count() * 1000  << " ms" << std::endl;
                    std::cout << "Total Time Chrono Launch with Graph Creation: " << totalLunchTimeWithChrono.count() * 1000  << " ms" << std::endl;
                    std::cout << "Total Time Chrono LunchExec without Graph Creation: " << totalTimeChrono.count() * 1000  << " ms" << std::endl;
                    std::cout << "Total Time Chrono LunchExec with Graph Creation: " << totalTimeWithChrono.count() * 1000 << " ms" << std::endl;
                    
                    totalTimeWithArr.push_back(totalTime + graphCreateTime);
                    totalTimeWithoutArr.push_back(totalTime);
                    chronoTotalTimeWithArr.push_back(totalTimeWithChrono.count() * 1000);
                    chronoTotalTimeWithoutArr.push_back(totalTimeChrono.count() * 1000);
                    chronoTotalLaunchTimeWithArr.push_back(totalLunchTimeWithChrono.count() * 1000);
                    chronoTotalLaunchTimeWithoutArr.push_back(totalLunchTimeChrono.count() * 1000);

                }
            }
        }
    }

    // Calculate mean and standard deviation
    // float meanTime = (totalTime + graphCreateTime) / NSTEP;
    // double varianceTime = (count > 1) ? M2 / (count - 1) : 0.0;
    // double stdDevTime = sqrt(varianceTime);

    // // Print out the time statistics
    // std::cout << "=======Setup (With Graph)=======" << std::endl;
    // std::cout << "Iterations: " << NSTEP << std::endl;
    // std::cout << "Skip By: " << SKIPBY << std::endl;
    // std::cout << "Kernels: kernelA, kernelB, kernelC" << std::endl;
    // std::cout << "Number of Blocks: " << numOfBlocks << std::endl;
    // std::cout << "Threads per Block: " << threadsPerBlock << std::endl;
    // std::cout << "Array Size: " << arraySize << std::endl;
    // std::cout << "=======Results (With Graph)=======" << std::endl;
    // std::cout << "Graph Creation Time: " << graphCreateTime << " ms" << std::endl;
    // std::cout << "Average Time with Graph: " << meanTime << " ms" << std::endl;
    // std::cout << "Average Time without Graph: " << (totalTime / (NSTEP - 1 - SKIPBY)) << " ms" << std::endl;
    // std::cout << "Variance: " << varianceTime << " ms^2" << std::endl;
    // std::cout << "Standard Deviation: " << stdDevTime << " ms" << std::endl;
    // std::cout << "Time Spread: " << lowerTime << " - " << upperTime << " ms" << std::endl;
    // std::cout << "Total Time without Graph Creation: " << totalTime << " ms" << std::endl;
    // std::cout << "Total Time with Graph Creation: " << totalTime + graphCreateTime << " ms" << std::endl;

    // Verify results
    constexpr double expected = initValue * 2.0 + 3;
    bool passed = true;
    for(size_t i = 0; i < arraySize; ++i){
        if(h_array[i] != expected){
            passed = false;
            std::cerr << "Validation failed! Index " << i << ": Expected " << expected << " got " << h_array[i] << std::endl;
            break;
        }
    }
    if(passed){
        std::cerr << "Validation passed." << std::endl;
    }

    // Free device memory asynchronously
    CUDA_CHECK(hipFreeAsync(d_arrayA, captureStream));
    CUDA_CHECK(hipFreeAsync(d_arrayB, captureStream));
    // Clean up
    CUDA_CHECK(hipEventDestroy(execStart));
    CUDA_CHECK(hipEventDestroy(execStop));
    CUDA_CHECK(hipEventDestroy(graphCreateStart));
    CUDA_CHECK(hipEventDestroy(graphCreateStop));
    CUDA_CHECK(hipGraphExecDestroy(graphExec));
    CUDA_CHECK(hipStreamDestroy(captureStream));
    CUDA_CHECK(hipHostFree(h_array));

    // Return total time including graph creation
    // return totalTime + graphCreateTime;
    // *totalTimeWith = totalTime + graphCreateTime;
    // *totalTimeWithout = totalTime;
}

int main(int argc, char* argv[]) {
    const int NSTEP = (argc > 1) ? atoi(argv[1]) : DEFAULT_NSTEP;
    const int SKIPBY = (argc > 2) ? atoi(argv[2]) : DEFAULT_SKIPBY;

    std::cout << "==============COMPLEX 3 DIFFERENT KERNELS TEST==============" << std::endl;
    
    std::vector<float> noneGraphTotalTimeWithArr, noneGraphTotalTimeWithoutArr;
    std::vector<float> chronoNoneGraphTotalTimeWithArr, chronoNoneGraphTotalTimeWithoutArr;
    std::vector<float> chronoNoneGraphTotalLaunchTimeWithArr, chronoNoneGraphTotalLaunchTimeWithoutArr;
    // Measure time for non-graph implementation
    // float nonGraphTotalTime, nonGraphTotalTimeWithout;
    // float nonGraphTotalTime = runWithoutGraph(N);
    // runWithoutGraph(&nonGraphTotalTime, &nonGraphTotalTimeWithout, NSTEP, SKIPBY);
    runWithoutGraph(noneGraphTotalTimeWithArr, noneGraphTotalTimeWithoutArr, 
                    chronoNoneGraphTotalTimeWithArr, chronoNoneGraphTotalTimeWithoutArr, 
                    chronoNoneGraphTotalLaunchTimeWithArr, chronoNoneGraphTotalLaunchTimeWithoutArr,
                    NSTEP, SKIPBY);

    std::vector<float> graphTotalTimeWithArr, graphTotalTimeWithoutArr;
    std::vector<float> chronoGraphTotalTimeWithArr, chronoGraphTotalTimeWithoutArr;
    std::vector<float> chronoGraphTotalLaunchTimeWithArr, chronoGraphTotalLaunchTimeWithoutArr;

    // Measure time for graph implementation
    // float graphTotalTime, graphTotalTimeWithout;
    // float graphTotalTime = runWithGraph(N);
    // runWithGraph(&graphTotalTime, &graphTotalTimeWithout, NSTEP, SKIPBY);
    runWithGraph(graphTotalTimeWithoutArr, graphTotalTimeWithArr, 
                chronoGraphTotalTimeWithArr, chronoGraphTotalTimeWithoutArr,
                chronoGraphTotalLaunchTimeWithArr, chronoGraphTotalLaunchTimeWithoutArr,
                NSTEP, SKIPBY);

    // // Compute the difference
    // float difference = nonGraphTotalTime - graphTotalTime;
    // float diffPerKernel = difference / (NSTEP);
    // float diffPercentage = (difference / nonGraphTotalTime) * 100;

    // // Compute the difference for without including Graph
    // float difference2 = nonGraphTotalTimeWithout - graphTotalTimeWithout;
    // float diffPerKernel2 = difference2 / (NSTEP-1);
    // float diffPercentage2 = (difference2 / nonGraphTotalTimeWithout) * 100;

    // // Print the differences
    // std::cout << "=======Comparison without Graph Creation=======" << std::endl;
    // std::cout << "Difference: " << difference2 << " ms" << std::endl;
    // std::cout << "Difference per step: " << diffPerKernel2 << " ms" << std::endl;
    // std::cout << "Difference percentage: " << diffPercentage2 << "%" << std::endl;

    // // Print the differences
    // std::cout << "=======Comparison=======" << std::endl;
    // std::cout << "Difference: " << difference << " ms" << std::endl;
    // std::cout << "Difference per step: " << diffPerKernel << " ms" << std::endl;
    // std::cout << "Difference percentage: " << diffPercentage << "%" << std::endl;

    // calculated
    // std::vector<float> diffTotalWithoutArr;
    // std::vector<float> diffPerStepWithoutArr;
    // std::vector<float> diffPercentWithoutArr;
    // std::vector<float> diffTotalWithArr;
    // std::vector<float> diffPerStepWithArr;
    // std::vector<float> diffPercentWithArr;

    std::vector<int> nsteps = generateSequence(NSTEP);

    for (int i = 0; i < nsteps.size(); i++) {

        // Compute the difference
        float difference = noneGraphTotalTimeWithArr[i] - graphTotalTimeWithArr[i];
        float diffPerKernel = difference / (nsteps[i] + 1);
        float diffPercentage = (difference / noneGraphTotalTimeWithArr[i]) * 100;

        // Compute the difference for without including Graph
        float difference2 = noneGraphTotalTimeWithoutArr[i] - graphTotalTimeWithoutArr[i];
        float diffPerKernel2 = difference2 / (nsteps[i]);
        float diffPercentage2 = (difference2 / noneGraphTotalTimeWithoutArr[i]) * 100;

        // Chrono Launch + Exec Time 
        float chronoDiffTotalTimeWith = chronoNoneGraphTotalTimeWithArr[i] - chronoGraphTotalTimeWithArr[i];
        float chronoDiffTotalTimeWithout = chronoNoneGraphTotalTimeWithoutArr[i] - chronoGraphTotalTimeWithoutArr[i];
        
        float chronoDiffPerStepWith = chronoDiffTotalTimeWith / (nsteps[i] + 1); 
        float chronoDiffPercentWith = (chronoDiffTotalTimeWith / chronoNoneGraphTotalTimeWithArr[i]) * 100;

        float chronoDiffPerStepWithout = chronoDiffTotalTimeWithout / (nsteps[i]); 
        float chronoDiffPercentWithout = (chronoDiffTotalTimeWithout / chronoNoneGraphTotalTimeWithoutArr[i]) * 100;

        // Chrono Launch Time
        float chronoDiffLaunchTimeWith = chronoNoneGraphTotalLaunchTimeWithArr[i] - chronoGraphTotalLaunchTimeWithArr[i];
        float chronoDiffLaunchTimeWithout = chronoNoneGraphTotalLaunchTimeWithoutArr[i] - chronoGraphTotalLaunchTimeWithoutArr[i];

        float chronoDiffLaunchPercentWithout = (chronoDiffLaunchTimeWithout / chronoNoneGraphTotalLaunchTimeWithoutArr[i]) * 100;
        float chronoDiffLaunchPercentWith = (chronoDiffLaunchTimeWith / chronoNoneGraphTotalLaunchTimeWithArr[i]) * 100;

        std::cout << "==============For NSTEP "<< nsteps[i] << "==============" << std::endl;
        // Print the differences
        std::cout << "=======Comparison without Graph Creation=======" << std::endl;
        std::cout << "Difference: " << difference2 << " ms" << std::endl;
        std::cout << "Difference per step: " << diffPerKernel2 << " ms" << std::endl;
        std::cout << "Difference percentage: " << diffPercentage2 << "%" << std::endl;

        // Print the differences
        std::cout << "=======Comparison=======" << std::endl;
        std::cout << "Difference: " << difference << " ms" << std::endl;
        std::cout << "Difference per step: " << diffPerKernel << " ms" << std::endl;
        std::cout << "Difference percentage: " << diffPercentage << "%" << std::endl;
        
        //----------------- Constructing CSV File------------------------------------
        // Make a new data entry with updated values
        CSVData newData;
        newData.NSTEP = nsteps[i];
        newData.SKIPBY = SKIPBY;
        newData.noneGraphTotalTimeWithout = noneGraphTotalTimeWithoutArr[i];
        newData.GraphTotalTimeWithout = graphTotalTimeWithoutArr[i];
        newData.noneGraphTotalTimeWith = noneGraphTotalTimeWithArr[i];
        newData.GraphTotalTimeWith = graphTotalTimeWithArr[i];
        newData.DiffTotalWithout = difference2;
        newData.DiffPerStepWithout = diffPerKernel2;
        newData.DiffPercentWithout = diffPercentage2;
        newData.DiffTotalWith = difference;
        newData.DiffPerStepWith = diffPerKernel;
        newData.DiffPercentWith = diffPercentage;
        newData.ChronoNoneGraphTotalTimeWithout = chronoNoneGraphTotalTimeWithoutArr[i];
        newData.ChronoGraphTotalTimeWithout = chronoGraphTotalTimeWithoutArr[i];
        newData.ChronoNoneGraphTotalLaunchTimeWithout = chronoNoneGraphTotalLaunchTimeWithoutArr[i];
        newData.ChronoGraphTotalLaunchTimeWithout = chronoGraphTotalLaunchTimeWithoutArr[i];
        newData.ChronoNoneGraphTotalTimeWith = chronoNoneGraphTotalTimeWithArr[i];
        newData.ChronoGraphTotalTimeWith = chronoGraphTotalTimeWithArr[i];
        newData.ChronoNoneGraphTotalLaunchTimeWith = chronoNoneGraphTotalLaunchTimeWithArr[i];
        newData.ChronoGraphTotalLaunchTimeWith = chronoGraphTotalLaunchTimeWithArr[i];
        newData.ChronoDiffTotalTimeWithout = chronoDiffTotalTimeWithout;
        newData.ChronoDiffPerStepWithout = chronoDiffPerStepWithout;
        newData.ChronoDiffPercentWithout = chronoDiffPercentWithout;
        newData.ChronoDiffTotalTimeWith = chronoDiffTotalTimeWith;
        newData.ChronoDiffPerStepWith = chronoDiffPerStepWith;
        newData.ChronoDiffPercentWith = chronoDiffPercentWith;
        newData.ChronoDiffLaunchTimeWithout = chronoDiffLaunchTimeWithout;
        newData.ChronoDiffLaunchPercentWithout = chronoDiffLaunchPercentWithout;
        newData.ChronoDiffLaunchTimeWith = chronoDiffLaunchTimeWith;
        newData.ChronoDiffLaunchPercentWith = chronoDiffLaunchPercentWith;

        // const std::string tempFILENAME = "complex_3_different_kernels.tmp";
        const std::string FILENAME = "complex_3_different_kernels.csv";

        updateOrAppendCSV(FILENAME, newData);
    }

    return 0;
}
