#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <cmath>    // For sqrt in standard deviation calculation

// Local headers
#include "../cuda_check.h"

#define NSTEP 1000
#define SKIPBY 0

// Kernel functions
__global__ void kernelA(double* arrayA, size_t size){
    const size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    if(x < size){ arrayA[x] *= 2.0; }
}

__global__ void kernelB(int* arrayB, size_t size){
    const size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    if(x < size){ arrayB[x] = 3; }
}

__global__ void kernelC(double* arrayA, const int* arrayB, size_t size){
    const size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    if(x < size){ arrayA[x] += arrayB[x]; }
}

struct set_vector_args {
    double* h_array;
    double value;
    size_t size;
};

void CUDART_CB set_vector(void* args) {
    set_vector_args* h_args = reinterpret_cast<set_vector_args*>(args);
    double* array = h_args->h_array;
    size_t size = h_args->size;
    double value = h_args->value;

    // Initialize h_array with the specified value
    for (size_t i = 0; i < size; ++i) {
        array[i] = value;
    }

    // Do NOT delete h_args here
}

// Function for non-graph implementation
void runWithoutGraph(float* totalTimeWith, float* totalTimeWithout) {
    constexpr int numOfBlocks = 1024;
    constexpr int threadsPerBlock = 1024;
    constexpr size_t arraySize = 1U << 20;
    // constexpr int iterations = 1000;
    constexpr double initValue = 2.0;

    // Host and device memory
    double* d_arrayA;
    int* d_arrayB;
    std::vector<double> h_array(arraySize);

    // Initialize host array using index i
    // for (size_t i = 0; i < arraySize; ++i) {
    //     h_array[i] = static_cast<double>(i);
    // }
    
    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_arrayA, arraySize * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_arrayB, arraySize * sizeof(int)));

    // Initialize host array
    h_array.assign(h_array.size(), initValue);

    // Set Timer for first run
    hipEvent_t firstCreateStart, firstCreateStop;
    float firstCreateTime = 0.0f;
    CUDA_CHECK(hipEventCreate(&firstCreateStart));
    CUDA_CHECK(hipEventCreate(&firstCreateStop));

    // Create a stream
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Start measuring first run time
    CUDA_CHECK(hipEventRecord(firstCreateStart, stream));

    // // Allocate device memory
    // CUDA_CHECK(hipMalloc(&d_arrayA, arraySize * sizeof(double)));
    // CUDA_CHECK(hipMalloc(&d_arrayB, arraySize * sizeof(int)));

    // // Initialize host array
    // h_array.assign(h_array.size(), initValue);

    // Copy h_array to device
    CUDA_CHECK(hipMemcpyAsync(d_arrayA, h_array.data(), arraySize * sizeof(double), hipMemcpyHostToDevice, stream));

    // Launch kernels
    kernelA<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayA, arraySize);
    kernelB<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayB, arraySize);
    kernelC<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayA, d_arrayB, arraySize);

    // Copy data back to host
    CUDA_CHECK(hipMemcpyAsync(h_array.data(), d_arrayA, arraySize * sizeof(double), hipMemcpyDeviceToHost, stream));

    // Wait for all operations to complete
    CUDA_CHECK(hipStreamSynchronize(stream));

    // Free device memory
    // CUDA_CHECK(hipFree(d_arrayA));
    // CUDA_CHECK(hipFree(d_arrayB));

    // Stop measuring first run time
    CUDA_CHECK(hipEventRecord(firstCreateStop, stream));
    CUDA_CHECK(hipEventSynchronize(firstCreateStop));
    CUDA_CHECK(hipEventElapsedTime(&firstCreateTime, firstCreateStart, firstCreateStop));

    // Measure execution time
    hipEvent_t execStart, execStop;
    CUDA_CHECK(hipEventCreate(&execStart));
    CUDA_CHECK(hipEventCreate(&execStop));

    float elapsedTime = 0.0f;
    float totalTime = 0.0f;
    float upperTime = 0.0f;
    float lowerTime = 0.0f;
    // int skipBy = 0;
    double mean = 0.0;
    double M2 = 0.0;
    int count = 0;

    // Execute the sequence multiple times
    for(int i = 0; i < NSTEP; ++i){

        // Allocate device memory
        CUDA_CHECK(hipMalloc(&d_arrayA, arraySize * sizeof(double)));
        CUDA_CHECK(hipMalloc(&d_arrayB, arraySize * sizeof(int)));

        // Initialize host array
        h_array.assign(h_array.size(), initValue);

        CUDA_CHECK(hipEventRecord(execStart, stream));

        // // Allocate device memory
        // CUDA_CHECK(hipMalloc(&d_arrayA, arraySize * sizeof(double)));
        // CUDA_CHECK(hipMalloc(&d_arrayB, arraySize * sizeof(int)));

        // // Initialize host array
        // h_array.assign(h_array.size(), initValue);

        // Copy h_array to device
        CUDA_CHECK(hipMemcpyAsync(d_arrayA, h_array.data(), arraySize * sizeof(double), hipMemcpyHostToDevice, stream));

        // Launch kernels
        kernelA<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayA, arraySize);
        kernelB<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayB, arraySize);
        kernelC<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayA, d_arrayB, arraySize);

        // Copy data back to host
        CUDA_CHECK(hipMemcpyAsync(h_array.data(), d_arrayA, arraySize * sizeof(double), hipMemcpyDeviceToHost, stream));

        // Wait for all operations to complete
        CUDA_CHECK(hipStreamSynchronize(stream));

        // Free device memory
        // CUDA_CHECK(hipFree(d_arrayA));
        // CUDA_CHECK(hipFree(d_arrayB));

        CUDA_CHECK(hipEventRecord(execStop, stream));
        CUDA_CHECK(hipEventSynchronize(execStop));
        CUDA_CHECK(hipEventElapsedTime(&elapsedTime, execStart, execStop));

        // Time calculations
        if (i >= SKIPBY) {
            totalTime += elapsedTime;

            // Welford's algorithm for calculating mean and variance
            count++;
            double delta = elapsedTime - mean;
            mean += delta / count;
            double delta2 = elapsedTime - mean;
            M2 += delta * delta2;

            if (elapsedTime > upperTime) {
                upperTime = elapsedTime;
            }
            if (elapsedTime < lowerTime || lowerTime == 0.0f) {
                lowerTime = elapsedTime;
            }
        }
    }

    // Calculate mean and standard deviation
    float meanTime = (totalTime + firstCreateTime) / NSTEP;
    double varianceTime = (count > 1) ? M2 / (count - 1) : 0.0;
    double stdDevTime = sqrt(varianceTime);

    // Print out the time statistics
    std::cout << "=======Setup (No Graph)=======" << std::endl;
    std::cout << "Iterations: " << NSTEP << std::endl;
    std::cout << "Skip By: " << SKIPBY << std::endl;
    std::cout << "Kernels: kernelA, kernelB, kernelC" << std::endl;
    std::cout << "Number of Blocks: " << numOfBlocks << std::endl;
    std::cout << "Threads per Block: " << threadsPerBlock << std::endl;
    std::cout << "Array Size: " << arraySize << std::endl;
    std::cout << "=======Results (No Graph)=======" << std::endl;
    std::cout << "First Run: " << firstCreateTime << " ms" << std::endl;
    std::cout << "Average Time with firstRun: " << meanTime << " ms" << std::endl;
    std::cout << "Average Time without firstRun: " << (totalTime / (NSTEP - 1 - SKIPBY)) << " ms" << std::endl;
    std::cout << "Variance: " << varianceTime << " ms^2" << std::endl;
    std::cout << "Standard Deviation: " << stdDevTime << " ms" << std::endl;
    std::cout << "Time Spread: " << lowerTime << " - " << upperTime << " ms" << std::endl;
    std::cout << "Total Time without firstRun: " << totalTime << " ms" << std::endl;
    std::cout << "Total Time with firstRun: " << totalTime + firstCreateTime << " ms" << std::endl;

    // Verify results
    constexpr double expected = initValue * 2.0 + 3;
    bool passed = true;
    for(size_t i = 0; i < arraySize; ++i){
        if(h_array[i] != expected){
            passed = false;
            std::cerr << "Validation failed! Expected " << expected << " got " << h_array[i] << " at index " << i << std::endl;
            break;
        }
    }
    if(passed){
        std::cerr << "Validation passed." << std::endl;
    }

    // Clean up
    CUDA_CHECK(hipEventDestroy(execStart));
    CUDA_CHECK(hipEventDestroy(execStop));
    CUDA_CHECK(hipEventDestroy(firstCreateStart));
    CUDA_CHECK(hipEventDestroy(firstCreateStop));
    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipFree(d_arrayA));
    CUDA_CHECK(hipFree(d_arrayB));

    // Return total time including first run
    // return totalTime + firstCreateTime;
    *totalTimeWith = totalTime + firstCreateTime;
    *totalTimeWithout = totalTime;
}

// Function for graph implementation
void runWithGraph(float* totalTimeWith, float* totalTimeWithout) {
    constexpr int numOfBlocks = 1024;
    constexpr int threadsPerBlock = 1024;
    constexpr size_t arraySize = 1U << 20;
    // constexpr int iterations = 1000;
    constexpr double initValue = 2.0;

    double* d_arrayA;
    int* d_arrayB;
    double* h_array = nullptr;
    CUDA_CHECK(hipHostMalloc((void**)&h_array, arraySize * sizeof(double)));

    // Initialize host array using index i
    // for (size_t i = 0; i < arraySize; ++i) {
    //     h_array[i] = static_cast<double>(i);
    // }

    // Set Timer for graph creation
    hipEvent_t graphCreateStart, graphCreateStop;
    float graphCreateTime = 0.0f;
    CUDA_CHECK(hipEventCreate(&graphCreateStart));
    CUDA_CHECK(hipEventCreate(&graphCreateStop));

    hipStream_t captureStream;
    CUDA_CHECK(hipStreamCreate(&captureStream));

    // Allocate device memory asynchronously
    CUDA_CHECK(hipMallocAsync(&d_arrayA, arraySize * sizeof(double), captureStream));
    CUDA_CHECK(hipMallocAsync(&d_arrayB, arraySize * sizeof(int), captureStream));

    set_vector_args* args = new set_vector_args{h_array, initValue, arraySize};
    CUDA_CHECK(hipLaunchHostFunc(captureStream, set_vector, args));

    // Start measuring graph creation time
    CUDA_CHECK(hipEventRecord(graphCreateStart, captureStream));

    // Start capturing operations
    CUDA_CHECK(hipStreamBeginCapture(captureStream, hipStreamCaptureModeGlobal));

    // // Allocate device memory asynchronously
    // CUDA_CHECK(hipMallocAsync(&d_arrayA, arraySize * sizeof(double), captureStream));
    // CUDA_CHECK(hipMallocAsync(&d_arrayB, arraySize * sizeof(int), captureStream));

    // set_vector_args* args = new set_vector_args{h_array, initValue, arraySize};
    // CUDA_CHECK(hipLaunchHostFunc(captureStream, set_vector, args));

    // Copy h_array to device
    CUDA_CHECK(hipMemcpyAsync(d_arrayA, h_array, arraySize * sizeof(double), hipMemcpyHostToDevice, captureStream));

    // Launch kernels
    kernelA<<<numOfBlocks, threadsPerBlock, 0, captureStream>>>(d_arrayA, arraySize);
    kernelB<<<numOfBlocks, threadsPerBlock, 0, captureStream>>>(d_arrayB, arraySize);
    kernelC<<<numOfBlocks, threadsPerBlock, 0, captureStream>>>(d_arrayA, d_arrayB, arraySize);

    // Copy data back to host
    CUDA_CHECK(hipMemcpyAsync(h_array, d_arrayA, arraySize * sizeof(double), hipMemcpyDeviceToHost, captureStream));

    // Free device memory asynchronously
    // CUDA_CHECK(hipFreeAsync(d_arrayA, captureStream));
    // CUDA_CHECK(hipFreeAsync(d_arrayB, captureStream));

    // Stop capturing
    hipGraph_t graph;
    CUDA_CHECK(hipStreamEndCapture(captureStream, &graph));

    // Create an executable graph
    hipGraphExec_t graphExec;
    CUDA_CHECK(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

    // Destroy the graph template if not needed
    CUDA_CHECK(hipGraphDestroy(graph));

    // Stop measuring graph creation time
    CUDA_CHECK(hipEventRecord(graphCreateStop, captureStream));
    CUDA_CHECK(hipEventSynchronize(graphCreateStop));
    CUDA_CHECK(hipEventElapsedTime(&graphCreateTime, graphCreateStart, graphCreateStop));

    // Measure execution time
    hipEvent_t execStart, execStop;
    CUDA_CHECK(hipEventCreate(&execStart));
    CUDA_CHECK(hipEventCreate(&execStop));

    float elapsedTime = 0.0f;
    float totalTime = 0.0f;
    float upperTime = 0.0f;
    float lowerTime = 0.0f;
    // int skipBy = 0;
    double mean = 0.0;
    double M2 = 0.0;
    int count = 0;

    // Launch the graph multiple times
    for(int i = 0; i < NSTEP; ++i){
        CUDA_CHECK(hipEventRecord(execStart, captureStream));

        CUDA_CHECK(hipGraphLaunch(graphExec, captureStream));
        CUDA_CHECK(hipStreamSynchronize(captureStream));

        CUDA_CHECK(hipEventRecord(execStop, captureStream));
        CUDA_CHECK(hipEventSynchronize(execStop));
        CUDA_CHECK(hipEventElapsedTime(&elapsedTime, execStart, execStop));

        // Time calculations
        if (i >= SKIPBY) {
            totalTime += elapsedTime;

            // Welford's algorithm for calculating mean and variance
            count++;
            double delta = elapsedTime - mean;
            mean += delta / count;
            double delta2 = elapsedTime - mean;
            M2 += delta * delta2;

            if (elapsedTime > upperTime) {
                upperTime = elapsedTime;
            }
            if (elapsedTime < lowerTime || lowerTime == 0.0f) {
                lowerTime = elapsedTime;
            }
        }
    }

    // Calculate mean and standard deviation
    float meanTime = (totalTime + graphCreateTime) / NSTEP;
    double varianceTime = (count > 1) ? M2 / (count - 1) : 0.0;
    double stdDevTime = sqrt(varianceTime);

    // Print out the time statistics
    std::cout << "=======Setup (With Graph)=======" << std::endl;
    std::cout << "Iterations: " << NSTEP << std::endl;
    std::cout << "Skip By: " << SKIPBY << std::endl;
    std::cout << "Kernels: kernelA, kernelB, kernelC" << std::endl;
    std::cout << "Number of Blocks: " << numOfBlocks << std::endl;
    std::cout << "Threads per Block: " << threadsPerBlock << std::endl;
    std::cout << "Array Size: " << arraySize << std::endl;
    std::cout << "=======Results (With Graph)=======" << std::endl;
    std::cout << "Graph Creation Time: " << graphCreateTime << " ms" << std::endl;
    std::cout << "Average Time with Graph: " << meanTime << " ms" << std::endl;
    std::cout << "Average Time without Graph: " << (totalTime / (NSTEP - 1 - SKIPBY)) << " ms" << std::endl;
    std::cout << "Variance: " << varianceTime << " ms^2" << std::endl;
    std::cout << "Standard Deviation: " << stdDevTime << " ms" << std::endl;
    std::cout << "Time Spread: " << lowerTime << " - " << upperTime << " ms" << std::endl;
    std::cout << "Total Time without Graph Creation: " << totalTime << " ms" << std::endl;
    std::cout << "Total Time with Graph Creation: " << totalTime + graphCreateTime << " ms" << std::endl;

    // Verify results
    constexpr double expected = initValue * 2.0 + 3;
    bool passed = true;
    for(size_t i = 0; i < arraySize; ++i){
        if(h_array[i] != expected){
            passed = false;
            std::cerr << "Validation failed! Index " << i << ": Expected " << expected << " got " << h_array[i] << std::endl;
            break;
        }
    }
    if(passed){
        std::cerr << "Validation passed." << std::endl;
    }

    // Clean up

    CUDA_CHECK(hipFreeAsync(d_arrayA, captureStream));
    CUDA_CHECK(hipFreeAsync(d_arrayB, captureStream));

    CUDA_CHECK(hipEventDestroy(execStart));
    CUDA_CHECK(hipEventDestroy(execStop));
    CUDA_CHECK(hipEventDestroy(graphCreateStart));
    CUDA_CHECK(hipEventDestroy(graphCreateStop));
    CUDA_CHECK(hipGraphExecDestroy(graphExec));
    CUDA_CHECK(hipStreamDestroy(captureStream));
    CUDA_CHECK(hipHostFree(h_array));

    // Return total time including graph creation
    // return totalTime + graphCreateTime;
    *totalTimeWith = totalTime + graphCreateTime;
    *totalTimeWithout = totalTime;
}

int main() {
    // Measure time for non-graph implementation
    // float nonGraphTotalTime = runWithoutGraph();

    // Measure time for graph implementation
    // float graphTotalTime = runWithGraph();

    // Measure time for non-graph implementation
    float nonGraphTotalTime, nonGraphTotalTimeWithout;
    // float nonGraphTotalTime = runWithoutGraph(N);
    runWithoutGraph(&nonGraphTotalTime, &nonGraphTotalTimeWithout);

    // Measure time for graph implementation
    float graphTotalTime, graphTotalTimeWithout;
    // float graphTotalTime = runWithGraph(N);
    runWithGraph(&graphTotalTime, &graphTotalTimeWithout);


    // Compute the difference
    float difference = nonGraphTotalTime - graphTotalTime;
    float diffPerKernel = difference / (NSTEP);
    float diffPercentage = (difference / nonGraphTotalTime) * 100;

    // Compute the difference for without including Graph
    float difference2 = nonGraphTotalTimeWithout - graphTotalTimeWithout;
    float diffPerKernel2 = difference2 / (NSTEP-1);
    float diffPercentage2 = (difference2 / nonGraphTotalTimeWithout) * 100;

    // Print the differences
    std::cout << "=======Comparison without Graph Creation=======" << std::endl;
    std::cout << "Difference: " << difference2 << " ms" << std::endl;
    std::cout << "Difference per step: " << diffPerKernel2 << " ms" << std::endl;
    std::cout << "Difference percentage: " << diffPercentage2 << "%" << std::endl;

    // Print the differences
    std::cout << "=======Comparison=======" << std::endl;
    std::cout << "Difference: " << difference << " ms" << std::endl;
    std::cout << "Difference per step: " << diffPerKernel << " ms" << std::endl;
    std::cout << "Difference percentage: " << diffPercentage << "%" << std::endl;

    return 0;
}
