#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

#define CUDA_CHECK(expression)                \
{                                             \
    const hipError_t status = expression;    \
    if(status != hipSuccess){                \
            std::cerr << "CUDA error "        \
                << status << ": "             \
                << hipGetErrorString(status) \
                << " at " << __FILE__ << ":"  \
                << __LINE__ << std::endl;     \
    }                                         \
}

__global__ void kernelA(double* arrayA, size_t size){
    const size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    if(x < size){ arrayA[x] *= 2.0; }
}

__global__ void kernelB(int* arrayB, size_t size){
    const size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    if(x < size){ arrayB[x] = 3; }
}

__global__ void kernelC(double* arrayA, const int* arrayB, size_t size){
    const size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    if(x < size){ arrayA[x] += arrayB[x]; }
}

int main(){
    constexpr int numOfBlocks = 1024;
    constexpr int threadsPerBlock = 1024;
    constexpr size_t arraySize = 1U << 20;

    double* d_arrayA;
    int* d_arrayB;
    std::vector<double> h_array(arraySize);
    constexpr double initValue = 2.0;

    // Set Timer for graph creation
    hipEvent_t firstCreateStart, firstCreateStop;
    float firstCreateTime = 0.0f;
    CUDA_CHECK(hipEventCreate(&firstCreateStart));
    CUDA_CHECK(hipEventCreate(&firstCreateStop));

    // Create a stream
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Start measuring graph creation time
    CUDA_CHECK(hipEventRecord(firstCreateStart, stream));

    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_arrayA, arraySize * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_arrayB, arraySize * sizeof(int)));

    // Initialize host array
    h_array.assign(h_array.size(), initValue);

    // Copy h_array to device
    CUDA_CHECK(hipMemcpyAsync(d_arrayA, h_array.data(), arraySize * sizeof(double), hipMemcpyHostToDevice, stream));

    // Launch kernels
    kernelA<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayA, arraySize);
    kernelB<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayB, arraySize);
    kernelC<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayA, d_arrayB, arraySize);

    // Copy data back to host
    CUDA_CHECK(hipMemcpyAsync(h_array.data(), d_arrayA, arraySize * sizeof(double), hipMemcpyDeviceToHost, stream));

    // Wait for all operations to complete
    CUDA_CHECK(hipStreamSynchronize(stream));

    // Free device memory
    CUDA_CHECK(hipFree(d_arrayA));
    CUDA_CHECK(hipFree(d_arrayB));

    // Stop measuring graph creation time
    CUDA_CHECK(hipEventRecord(firstCreateStop, stream));
    CUDA_CHECK(hipEventSynchronize(firstCreateStop));
    CUDA_CHECK(hipEventElapsedTime(&firstCreateTime, firstCreateStart, firstCreateStop));

    // Measure execution time
    hipEvent_t execStart, execStop;
    float execTime = 0.0f;
    CUDA_CHECK(hipEventCreate(&execStart));
    CUDA_CHECK(hipEventCreate(&execStop));

    CUDA_CHECK(hipEventRecord(execStart, stream));

    // Execute the sequence multiple times
    constexpr int iterations = 1000;
    for(int i = 0; i < iterations; ++i){
        // Allocate device memory
        CUDA_CHECK(hipMalloc(&d_arrayA, arraySize * sizeof(double)));
        CUDA_CHECK(hipMalloc(&d_arrayB, arraySize * sizeof(int)));

        // Initialize host array
        h_array.assign(h_array.size(), initValue);

        // Copy h_array to device
        CUDA_CHECK(hipMemcpyAsync(d_arrayA, h_array.data(), arraySize * sizeof(double), hipMemcpyHostToDevice, stream));

        // Launch kernels
        kernelA<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayA, arraySize);
        kernelB<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayB, arraySize);
        kernelC<<<numOfBlocks, threadsPerBlock, 0, stream>>>(d_arrayA, d_arrayB, arraySize);

        // Copy data back to host
        CUDA_CHECK(hipMemcpyAsync(h_array.data(), d_arrayA, arraySize * sizeof(double), hipMemcpyDeviceToHost, stream));

        // Wait for all operations to complete
        CUDA_CHECK(hipStreamSynchronize(stream));

        // Free device memory
        CUDA_CHECK(hipFree(d_arrayA));
        CUDA_CHECK(hipFree(d_arrayB));
    }

    CUDA_CHECK(hipEventRecord(execStop, stream));
    CUDA_CHECK(hipEventSynchronize(execStop));
    CUDA_CHECK(hipEventElapsedTime(&execTime, execStart, execStop));

    std::cout << "First Run: " << firstCreateTime << "ms" << std::endl;
    std::cout << "Iterations: " << iterations << std::endl;
    std::cout << "Average Execution Time per Iteration: " << (execTime / iterations) << "ms" << std::endl;
    std::cout << "Total Time: " << execTime + firstCreateTime << "ms" << std::endl;
    std::cout << "New Average Execution Time per Iteration: " << ((execTime + firstCreateTime) / (iterations + 1)) << "ms" << std::endl;

    // Verify results
    constexpr double expected = initValue * 2.0 + 3;
    bool passed = true;
    for(size_t i = 0; i < arraySize; ++i){
        if(h_array[i] != expected){
            passed = false;
            std::cerr << "Validation failed! Expected " << expected << " got " << h_array[i] << " at index " << i << std::endl;
            break;
        }
    }
    if(passed){
        std::cerr << "Validation passed." << std::endl;
    }

    // Clean up
    CUDA_CHECK(hipEventDestroy(execStart));
    CUDA_CHECK(hipEventDestroy(execStop));
    CUDA_CHECK(hipEventDestroy(firstCreateStart));
    CUDA_CHECK(hipEventDestroy(firstCreateStop));
    CUDA_CHECK(hipStreamDestroy(stream));

    return 0;
}
