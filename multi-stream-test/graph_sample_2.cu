#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

#define CUDA_CHECK(expression)                \
{                                             \
    const hipError_t status = expression;    \
    if(status != hipSuccess){                \
        std::cerr << "CUDA error "            \
                  << status << ": "           \
                  << hipGetErrorString(status) \
                  << " at " << __FILE__ << ":"  \
                  << __LINE__ << std::endl;     \
    }                                         \
}

__global__ void kernelA(double* arrayA, size_t size){
    const size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    if(x < size){ arrayA[x] *= 2.0; }
}

__global__ void kernelB(int* arrayB, size_t size){
    const size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    if(x < size){ arrayB[x] = 3; }
}

__global__ void kernelC(double* arrayA, const int* arrayB, size_t size){
    const size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    if(x < size){ arrayA[x] += arrayB[x]; }
}

// struct set_vector_args{
//     // std::vector<double>* h_array;
//     std::vector<double>& h_array;
//     double value;
// };

struct set_vector_args {
    double* h_array;
    double value;
    size_t size;
};

void CUDART_CB set_vector(void* args) {
    set_vector_args* h_args = reinterpret_cast<set_vector_args*>(args);
    double* array = h_args->h_array;
    size_t size = h_args->size;
    double value = h_args->value;

    // Initialize h_array with the specified value
    for (size_t i = 0; i < size; ++i) {
        array[i] = value;
    }

    // Do NOT delete h_args here
}


// void CUDART_CB set_vector(void* args) {
//     // attempt 1
//     // set_vector_args h_args{*(reinterpret_cast<set_vector_args*>(args))};

//     // std::vector<double>& vec{h_args.h_array};
//     // vec.assign(vec.size(), h_args.value);
//     //attempt 2
//     // set_vector_args* h_args = reinterpret_cast<set_vector_args*>(args);
//     // std::vector<double>* vec = h_args->h_array;
//     // vec->assign(vec->size(), h_args->value);
//     // delete h_args;  // Free the dynamically allocated memory
//     // attempt 3
//     set_vector_args* h_args = reinterpret_cast<set_vector_args*>(args);
//     std::vector<double>& vec = h_args->h_array;
//     vec.assign(vec.size(), h_args->value);
//     // delete h_args;  // Free the dynamically allocated memory
// }

int main(){
    constexpr int numOfBlocks = 1024;
    constexpr int threadsPerBlock = 1024;
    constexpr size_t arraySize = 1U << 20;

    double* d_arrayA;
    int* d_arrayB;
    // std::vector<double> h_array(arraySize);
    double* h_array = nullptr;
    CUDA_CHECK(hipHostMalloc((void**)&h_array, arraySize * sizeof(double), hipHostMallocDefault));

    constexpr double initValue = 2.0;

    // Set Timer for graph creation
    hipEvent_t graphCreateStart, graphCreateStop;
    float graphCreateTime = 0.0f;
    CUDA_CHECK(hipEventCreate(&graphCreateStart));
    CUDA_CHECK(hipEventCreate(&graphCreateStop));

    hipStream_t captureStream;
    CUDA_CHECK(hipStreamCreate(&captureStream));

    // Start measuring graph creation time
    CUDA_CHECK(hipEventRecord(graphCreateStart, captureStream));

    // ##### Start capturing operations
    CUDA_CHECK(hipStreamBeginCapture(captureStream, hipStreamCaptureModeGlobal));

    // cudaMallocAsync and cudaMemcpyAsync are needed, to be able to assign it to a stream
    CUDA_CHECK(hipMallocAsync(&d_arrayA, arraySize * sizeof(double), captureStream));
    CUDA_CHECK(hipMallocAsync(&d_arrayB, arraySize * sizeof(int), captureStream));

    // Assign host function to the stream

    // Attempt 1: Needs a custom struct to pass the arguments
    // set_vector_args args{h_array, initValue};
    // CUDA_CHECK(cudaLaunchHostFunc(captureStream, set_vector, &args));

    // Attempt 2: Dynamically allocate args to ensure it remains valid
    // set_vector_args* args = new set_vector_args{&h_array, initValue};
    // CUDA_CHECK(cudaLaunchHostFunc(captureStream, set_vector, args));

    // Attempt 3: Dynamically allocate args to ensure it remains valid
    // set_vector_args* args = new set_vector_args{h_array, initValue};
    // CUDA_CHECK(cudaLaunchHostFunc(captureStream, set_vector, args));
    set_vector_args* args = new set_vector_args{h_array, initValue, arraySize};
    CUDA_CHECK(hipLaunchHostFunc(captureStream, set_vector, args));


    CUDA_CHECK(hipMemcpyAsync(d_arrayA, h_array, arraySize * sizeof(double), hipMemcpyHostToDevice, captureStream));

    kernelA<<<numOfBlocks, threadsPerBlock, 0, captureStream>>>(d_arrayA, arraySize);
    kernelB<<<numOfBlocks, threadsPerBlock, 0, captureStream>>>(d_arrayB, arraySize);
    kernelC<<<numOfBlocks, threadsPerBlock, 0, captureStream>>>(d_arrayA, d_arrayB, arraySize);

    CUDA_CHECK(hipMemcpyAsync(h_array, d_arrayA, arraySize * sizeof(*d_arrayA), hipMemcpyDeviceToHost, captureStream));

    CUDA_CHECK(hipFreeAsync(d_arrayA, captureStream));
    CUDA_CHECK(hipFreeAsync(d_arrayB, captureStream));

    // ###### Stop capturing
    hipGraph_t graph;
    CUDA_CHECK(hipStreamEndCapture(captureStream, &graph));

    // Create an executable graph
    hipGraphExec_t graphExec;
    CUDA_CHECK(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

    // Destroy the graph template if not needed
    CUDA_CHECK(hipGraphDestroy(graph));

    // Stop measuring graph creation time
    CUDA_CHECK(hipEventRecord(graphCreateStop, captureStream));
    CUDA_CHECK(hipEventSynchronize(graphCreateStop));
    CUDA_CHECK(hipEventElapsedTime(&graphCreateTime, graphCreateStart, graphCreateStop));

    // Now measure the execution time separately
    hipEvent_t execStart, execStop;
    float execTime = 0.0f;
    CUDA_CHECK(hipEventCreate(&execStart));
    CUDA_CHECK(hipEventCreate(&execStop));

    CUDA_CHECK(hipEventRecord(execStart, captureStream));

    // Launch the graph multiple times
    constexpr int iterations = 1000;
    for(int i = 0; i < iterations; ++i){
        CUDA_CHECK(hipGraphLaunch(graphExec, captureStream));
        CUDA_CHECK(hipStreamSynchronize(captureStream));
    }

    // CUDA_CHECK(cudaStreamSynchronize(captureStream));
    CUDA_CHECK(hipEventRecord(execStop, captureStream));
    CUDA_CHECK(hipEventSynchronize(execStop));
    CUDA_CHECK(hipEventElapsedTime(&execTime, execStart, execStop));

    std::cout << "Graph Creation Time: " << graphCreateTime << "ms" << std::endl;
    std::cout << "Iterations: " << iterations << std::endl;
    std::cout << "Average Execution Time per Iteration: " << (execTime / iterations) << "ms" << std::endl;
    std::cout << "Total Time: " << graphCreateTime + execTime << "ms" << std::endl;
    std::cout << "Average Execution Time per Iteration: " << ((execTime + graphCreateTime) / (iterations)) << "ms" << std::endl;

    // Verify results
    constexpr double expected = initValue * 2.0 + 3;
    bool passed = true;
    for(size_t i = 0; i < arraySize; ++i){
        if(h_array[i] != expected){
            passed = false;
            std::cerr << "Validation failed! Index " << i << ": Expected " << expected << " got " << h_array[i] << std::endl;
            break;
        }
    }
    if(passed){
        std::cerr << "Validation passed."<< expected << " : " << h_array[arraySize - 1] << std::endl;
    }

    CUDA_CHECK(hipEventDestroy(execStart));
    CUDA_CHECK(hipEventDestroy(execStop));
    CUDA_CHECK(hipEventDestroy(graphCreateStart));
    CUDA_CHECK(hipEventDestroy(graphCreateStop));
    // Free graph and stream resources after usage
    CUDA_CHECK(hipGraphExecDestroy(graphExec));
    CUDA_CHECK(hipStreamDestroy(captureStream));
    delete args;
    CUDA_CHECK(hipHostFree(h_array));


    return 0;
}
