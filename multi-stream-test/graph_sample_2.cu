#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

#define CUDA_CHECK(expression)                \
{                                             \
    const hipError_t status = expression;    \
    if(status != hipSuccess){                \
            std::cerr << "CUDA error "        \
                << status << ": "             \
                << hipGetErrorString(status) \
                << " at " << __FILE__ << ":"  \
                << __LINE__ << std::endl;     \
    }                                         \
}

__global__ void kernelA(double* arrayA, size_t size){
    const size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    if(x < size){arrayA[x] *= 2.0;}
}

__global__ void kernelB(int* arrayB, size_t size){
    const size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    if(x < size){arrayB[x] = 3;}
}

__global__ void kernelC(double* arrayA, const int* arrayB, size_t size){
    const size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    if(x < size){arrayA[x] += arrayB[x];}
}

struct set_vector_args{
    std::vector<double>& h_array;
    double value;
};

void CUDART_CB set_vector(void* args){
    set_vector_args h_args{*(reinterpret_cast<set_vector_args*>(args))};

    std::vector<double>& vec{h_args.h_array};
    vec.assign(vec.size(), h_args.value);
}

int main(){
    constexpr int numOfBlocks = 1024;
    constexpr int threadsPerBlock = 1024;
    constexpr size_t arraySize = 1U << 20;

    // This example assumes that kernelA operates on data that needs to be initialized on
    // and copied from the host, while kernelB initializes the array that is passed to it.
    // Both arrays are then used as input to kernelC, where arrayA is also used as
    // output, that is copied back to the host, while arrayB is only read from and not modified.

    double* d_arrayA;
    int* d_arrayB;
    std::vector<double> h_array(arraySize);
    constexpr double initValue = 2.0;

    // Set Timer for graph creation
    hipEvent_t graphCreateStart, graphCreateStop;
    float graphCreateTime = 0.0f;
    CUDA_CHECK(hipEventCreate(&graphCreateStart));
    CUDA_CHECK(hipEventCreate(&graphCreateStop));

    hipStream_t captureStream;
    CUDA_CHECK(hipStreamCreate(&captureStream));

    // Start measuring graph creation time
    CUDA_CHECK(hipEventRecord(graphCreateStart, captureStream));

    // ##### Start capturing operations
    CUDA_CHECK(hipStreamBeginCapture(captureStream, hipStreamCaptureModeGlobal));

    // cudaMallocAsync and cudaMemcpyAsync are needed, to be able to assign it to a stream
    CUDA_CHECK(hipMallocAsync(&d_arrayA, arraySize*sizeof(double), captureStream));
    CUDA_CHECK(hipMallocAsync(&d_arrayB, arraySize*sizeof(int), captureStream));

    // Assign host function to the stream
    // Needs a custom struct to pass the arguments
    set_vector_args args{h_array, initValue};
    CUDA_CHECK(hipLaunchHostFunc(captureStream, set_vector, &args));

    CUDA_CHECK(hipMemcpyAsync(d_arrayA, h_array.data(), arraySize*sizeof(double), hipMemcpyHostToDevice, captureStream));

    kernelA<<<numOfBlocks, threadsPerBlock, 0, captureStream>>>(d_arrayA, arraySize);
    kernelB<<<numOfBlocks, threadsPerBlock, 0, captureStream>>>(d_arrayB, arraySize);
    kernelC<<<numOfBlocks, threadsPerBlock, 0, captureStream>>>(d_arrayA, d_arrayB, arraySize);

    CUDA_CHECK(hipMemcpyAsync(h_array.data(), d_arrayA, arraySize*sizeof(*d_arrayA), hipMemcpyDeviceToHost, captureStream));

    CUDA_CHECK(hipFreeAsync(d_arrayA, captureStream));
    CUDA_CHECK(hipFreeAsync(d_arrayB, captureStream));

    // ###### Stop capturing
    hipGraph_t graph;
    CUDA_CHECK(hipStreamEndCapture(captureStream, &graph));

    // Create an executable graph
    hipGraphExec_t graphExec;
    CUDA_CHECK(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

    // Destroy the graph template if not needed
    CUDA_CHECK(hipGraphDestroy(graph));

    // Stop measuring graph creation time
    CUDA_CHECK(hipEventRecord(graphCreateStop, captureStream));
    CUDA_CHECK(hipEventSynchronize(graphCreateStop));
    CUDA_CHECK(hipEventElapsedTime(&graphCreateTime, graphCreateStart, graphCreateStop));

    // Now measure the execution time separately
    hipEvent_t execStart, execStop;
    float execTime = 0.0f;
    CUDA_CHECK(hipEventCreate(&execStart));
    CUDA_CHECK(hipEventCreate(&execStop));

    CUDA_CHECK(hipEventRecord(execStart, captureStream));

    // Launch the graph multiple times
    constexpr int iterations = 1000;
    for(int i = 0; i < iterations; ++i){
        CUDA_CHECK(hipGraphLaunch(graphExec, captureStream));
    }

    CUDA_CHECK(hipEventRecord(execStop, captureStream));
    CUDA_CHECK(hipEventSynchronize(execStop));
    CUDA_CHECK(hipEventElapsedTime(&execTime, execStart, execStop));

    std::cout << "Graph Creation Time: " << graphCreateTime << "ms" << std::endl;
    std::cout << "Iterations: " << iterations << std::endl;
    std::cout << "Average Execution Time per Iteration: " << (execTime / iterations) << "ms" << std::endl;
    std::cout << "Total Time: " << graphCreateTime + execTime << "ms" << std::endl;
    std::cout << "Average Execution Time per Iteration: " << ((execTime + graphCreateTime) / (iterations + 1)) << "ms" << std::endl;

    // Verify results
    constexpr double expected = initValue * 2.0 + 3;
    bool passed = true;
    for(size_t i = 0; i < arraySize; ++i){
            if(h_array[i] != expected){
                    passed = false;
                    std::cerr << "Validation failed! Expected " << expected << " got " << h_array[0] << std::endl;
                    break;
            }
    }
    if(passed){
            std::cerr << "Validation passed." << std::endl;
    }

    CUDA_CHECK(hipEventDestroy(execStart));
    CUDA_CHECK(hipEventDestroy(execStop));
    CUDA_CHECK(hipEventDestroy(graphCreateStart));
    CUDA_CHECK(hipEventDestroy(graphCreateStop));
    // Free graph and stream resources after usage
    CUDA_CHECK(hipGraphExecDestroy(graphExec));
    CUDA_CHECK(hipStreamDestroy(captureStream));

    return 0;
}
