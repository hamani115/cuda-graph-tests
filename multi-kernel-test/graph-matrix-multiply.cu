#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include <iostream>

#include "cuda_check.h"

#define N 64//(1<<6) // Matrix dimensions (4096x4096)

#define NSTEP 100000//10
#define NKERNEL 10

__global__ void matMulKernel(float* A, float* B, float* C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    if (row < width && col < width) {
        for (int i = 0; i < width; i++) {
            sum += A[row * width + i] * B[i * width + col];
        }
        C[row * width + col] = sum;
    }
}

void matrixMultiplyWithGraph(float* A, float* B, float* C, int width) {
    dim3 block(32, 32);
    // dim3 grid((width + block.x - 1) / block.x, (width + block.y - 1) / block.y); //()im
    dim3 grid(6,6);

    hipStream_t stream;
    hipStreamCreate(&stream);

    // Create the CUDA graph
    hipGraph_t graph;
    hipGraphExec_t graphExec;
    hipEvent_t start, stop;
    float elapsedTime = 0.0f;
    float graphCreateTime = 0.0f;
    float totalTime = 0.0f; 
    float upperTime = 0.0f;
    float lowerTime = 0.0f; 
    int skipBy = 0;  
    hipEventCreate(&start); 
    hipEventCreate(&stop); 

    CUDA_CHECK(hipEventRecord(start, stream)); 
    // Begin graph capture
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    
    for (int i = 0; i < NKERNEL; i++) {  // Run 100 iterations
        matMulKernel<<<grid, block, 0, stream>>>(A, B, C, width);
    }

    // End graph capture
    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);

    CUDA_CHECK(hipEventRecord(stop, stream));
    CUDA_CHECK(hipEventSynchronize(stop)); 
    CUDA_CHECK(hipEventElapsedTime(&graphCreateTime, start, stop)); 

    for (int i = 0; i < NSTEP-1; i++) {
        //
        CUDA_CHECK(hipEventRecord(start, stream));  
        // Launch the graph
        hipGraphLaunch(graphExec, stream);
        hipStreamSynchronize(stream); // Ensure all kernels finish
        // 
        CUDA_CHECK(hipEventRecord(stop, stream));
        CUDA_CHECK(hipEventSynchronize(stop)); 
        CUDA_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));  
        if(i >= skipBy){
            totalTime += elapsedTime;  
            if(elapsedTime > upperTime) { 
                upperTime = elapsedTime; 
            } 
            if(elapsedTime < lowerTime) { 
                lowerTime = elapsedTime; 
            }  
            if(i == skipBy){ 
                lowerTime = elapsedTime; 
            } 
        }
    }
    float AverageTime = (totalTime + graphCreateTime) / (NSTEP - skipBy);
    std::cout << "Average Time: " << AverageTime << "ms" << std::endl;
    std::cout << "Time Spread: " << upperTime <<  " - " << lowerTime << "ms" << std::endl;
    std::cout << "Total Time without Graph Create: " << totalTime << "ms" << std::endl;
    std::cout << "Total Time with Graph Create: " << totalTime + graphCreateTime << "ms" << std::endl;
    // Cleanup
    hipGraphDestroy(graph);
    hipGraphExecDestroy(graphExec);
    hipStreamDestroy(stream);
}

int main() {
    // Allocate host memory
    float* h_A = (float*)malloc(N * N * sizeof(float));
    float* h_B = (float*)malloc(N * N * sizeof(float));
    float* h_C = (float*)malloc(N * N * sizeof(float));

    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = rand() % 100;
        h_B[i] = rand() % 100;
    }

    // Allocate device memory
    float* d_A, * d_B, * d_C;
    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));

    // Copy matrices to device
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Measure time
    auto start = std::chrono::high_resolution_clock::now();
    matrixMultiplyWithGraph(d_A, d_B, d_C, N);
    auto end = std::chrono::high_resolution_clock::now();

    // Copy result back to host
    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Calculate elapsed time
    std::chrono::duration<double> elapsed = end - start;
    printf("Elapsed time with CUDA Graphs: %f seconds\n", elapsed.count());

    // Cleanup
    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}

