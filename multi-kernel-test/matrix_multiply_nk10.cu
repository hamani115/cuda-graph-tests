#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
// #include <chrono>
#include <iostream>

// Local headers
#include "../cuda_check.h"

#define N 64  // Matrix dimensions (64x64)

#define NSTEP 100000
#define NKERNEL 10  // INDEPENDENT VARIABLE: CHANGE THE NUMBER OF KERNELS (10 OR 100)

// CUDA kernel for matrix multiplication
__global__ void matMulKernel(float* A, float* B, float* C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    if (row < width && col < width) {
        for (int i = 0; i < width; i++) {
            sum += A[row * width + i] * B[i * width + col];
        }
        C[row * width + col] = sum;
    }
}

// Function to perform matrix multiplication without using CUDA Graphs
void matrixMultiplyNoGraph(float* A, float* B, float* C, int width) {
    // Define block and grid sizes
    dim3 block(32, 32);  // 1024 threads
    // dim3 grid((width + block.x - 1) / block.x, (width + block.y - 1) / block.y);
    dim3 grid(6, 6);  // 36 Blocks

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Create CUDA events
    hipEvent_t start, stop;
    float elapsedTime = 0.0f;
    float firstTime = 0.0f;
    float totalTime = 0.0f;
    float upperTime = 0.0f;
    float lowerTime = 0.0f;
    int skipBy = 0;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Start recording time for first run
    CUDA_CHECK(hipEventRecord(start, stream));

    // Begin first run
    for (int i = 0; i < NKERNEL; i++) {  // Run NKERNEL iterations
        matMulKernel<<<grid, block, 0, stream>>>(A, B, C, width);
    }
    CUDA_CHECK(hipGetLastError());  // Check for kernel launch errors
    CUDA_CHECK(hipStreamSynchronize(stream));  // Ensure all kernels finish

    // Stop recording time for first run
    CUDA_CHECK(hipEventRecord(stop, stream));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&firstTime, start, stop));

    // Execute the kernel multiple times and measure performance
    for (int j = 0; j < NSTEP - 1; j++) {
        // Start the timer for each iteration
        CUDA_CHECK(hipEventRecord(start, stream));

        // Launch the kernel multiple times
        for (int i = 0; i < NKERNEL; i++) {  // Run NKERNEL iterations
            matMulKernel<<<grid, block, 0, stream>>>(A, B, C, width);
        }
        CUDA_CHECK(hipGetLastError());  // Check for kernel launch errors
        CUDA_CHECK(hipStreamSynchronize(stream));  // Ensure all kernels finish

        // Stop the timer for each iteration
        CUDA_CHECK(hipEventRecord(stop, stream));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));

        // Time calculations
        if (j >= skipBy) {
            totalTime += elapsedTime;
            if (elapsedTime > upperTime) {
                upperTime = elapsedTime;
            }
            if (elapsedTime < lowerTime) {
                lowerTime = elapsedTime;
            }
            if (istep == skipBy) {
                lowerTime = elapsedTime;
            }
        }
    }

    // Print time statistics
    float averageTime = (totalTime + firstTime) / (NSTEP - skipBy);
    std::cout << "Average Time: " << averageTime << " ms" << std::endl;
    std::cout << "Time Spread: " << upperTime << " - " << lowerTime << " ms" << std::endl;
    std::cout << "Total Time without first run: " << totalTime << " ms" << std::endl;
    std::cout << "Total Time with first run: " << (totalTime + firstTime) << " ms" << std::endl;

    // Destroy the CUDA stream and events
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}

int main() {
    // Allocate host memory
    float* h_A = (float*)malloc(N * N * sizeof(float));
    float* h_B = (float*)malloc(N * N * sizeof(float));
    float* h_C = (float*)malloc(N * N * sizeof(float));

    // Check host memory allocation
    if (!h_A || !h_B || !h_C) {
        fprintf(stderr, "Failed to allocate host memory\n");
        return EXIT_FAILURE;
    }

    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = static_cast<float>(rand() % 100);
        h_B[i] = static_cast<float>(rand() % 100);
    }

    // Allocate device memory
    float* d_A;
    float* d_B;
    float* d_C;
    CUDA_CHECK(hipMalloc(&d_A, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_B, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_C, N * N * sizeof(float)));

    // Copy matrices to device
    CUDA_CHECK(hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice));

    // Measure time using std::chrono
    // auto start = std::chrono::high_resolution_clock::now();
    // matrixMultiplyNoGraph(d_A, d_B, d_C, N);
    // auto end = std::chrono::high_resolution_clock::now();

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost));

    // Calculate elapsed time
    // std::chrono::duration<double> elapsed = end - start;
    // printf("Elapsed time without CUDA Graphs: %f seconds\n", elapsed.count());

    // Cleanup
    free(h_A);
    free(h_B);
    free(h_C);
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    return 0;
}
