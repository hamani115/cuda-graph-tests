#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include "cuda_check.h"

#define N (1<<12)//1024  // Size of the arrays

#define NSTEP 10000//2000
//#define NKERNEL 1

// CUDA kernel to add 10 arrays element-wise
__global__ void add_arrays(float *a1, float *a2, float *a3, float *a4, float *a5,
                           float *a6, float *a7, float *a8, float *a9, float *a10,
                           float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        result[i] = a1[i] + a2[i] + a3[i] + a4[i] + a5[i]
                  + a6[i] + a7[i] + a8[i] + a9[i] + a10[i];
    }
}

int main() {
    const int size = N * sizeof(float);

    // Allocate host memory
    float *h_a1 = (float*)malloc(size);
    float *h_a2 = (float*)malloc(size);
    float *h_a3 = (float*)malloc(size);
    float *h_a4 = (float*)malloc(size);
    float *h_a5 = (float*)malloc(size);
    float *h_a6 = (float*)malloc(size);
    float *h_a7 = (float*)malloc(size);
    float *h_a8 = (float*)malloc(size);
    float *h_a9 = (float*)malloc(size);
    float *h_a10 = (float*)malloc(size);
    float *h_result = (float*)malloc(size);

    // Initialize host arrays with values
    for (int i = 0; i < N; i++) {
        h_a1[i] = 1.0f;
        h_a2[i] = 2.0f;
        h_a3[i] = 3.0f;
        h_a4[i] = 4.0f;
        h_a5[i] = 5.0f;
        h_a6[i] = 6.0f;
        h_a7[i] = 7.0f;
        h_a8[i] = 8.0f;
        h_a9[i] = 9.0f;
        h_a10[i] = 10.0f;
        h_result[i] = 0.0f;  // Initialize result array to zero
    }

    // Allocate device memory
    float *d_a1, *d_a2, *d_a3, *d_a4, *d_a5;
    float *d_a6, *d_a7, *d_a8, *d_a9, *d_a10;
    float *d_result;

    hipMalloc((void**)&d_a1, size);
    hipMalloc((void**)&d_a2, size);
    hipMalloc((void**)&d_a3, size);
    hipMalloc((void**)&d_a4, size);
    hipMalloc((void**)&d_a5, size);
    hipMalloc((void**)&d_a6, size);
    hipMalloc((void**)&d_a7, size);
    hipMalloc((void**)&d_a8, size);
    hipMalloc((void**)&d_a9, size);
    hipMalloc((void**)&d_a10, size);
    hipMalloc((void**)&d_result, size);

    // Set Timer
    hipEvent_t start, stop;
    float elapsedTime = 0.0f;
    float graphCreateTime = 0.0f;
    float totalTime = 0.0f;
    float upperTime = 0.0f;
    float lowerTime = 0.0f;
    int skipBy = 100;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Create a CUDA stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Define execution configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // hipMallocAsync((void**)&d_a1, size, stream);
    // hipMallocAsync((void**)&d_a2, size, stream);
    // hipMallocAsync((void**)&d_a3, size, stream);
    // hipMallocAsync((void**)&d_a4, size, stream);
    // hipMallocAsync((void**)&d_a5, size, stream);
    // hipMallocAsync((void**)&d_a6, size, stream);
    // hipMallocAsync((void**)&d_a7, size, stream);
    // hipMallocAsync((void**)&d_a8, size, stream);
    // hipMallocAsync((void**)&d_a9, size, stream);
    // hipMallocAsync((void**)&d_a10, size, stream);
    // hipMallocAsync((void**)&d_result, size, stream);
    CUDA_CHECK(hipEventRecord(start, stream));
    //Graph
    hipGraph_t graph;
    hipGraphExec_t instance;
    CUDA_CHECK(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));

    // Copy host arrays to device arrays asynchronously
    CUDA_CHECK(hipMemcpyAsync(d_a1, h_a1, size, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_a2, h_a2, size, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_a3, h_a3, size, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_a4, h_a4, size, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_a5, h_a5, size, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_a6, h_a6, size, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_a7, h_a7, size, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_a8, h_a8, size, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_a9, h_a9, size, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_a10, h_a10, size, hipMemcpyHostToDevice, stream));

    // Launch kernel to add arrays on the created stream
    add_arrays<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        d_a1, d_a2, d_a3, d_a4, d_a5,
        d_a6, d_a7, d_a8, d_a9, d_a10,
        d_result
    );

    // Copy result back to host asynchronously
    CUDA_CHECK(hipMemcpyAsync(h_result, d_result, size, hipMemcpyDeviceToHost, stream));
    
    // End Capture
    CUDA_CHECK(hipStreamEndCapture(stream, &graph));
    CUDA_CHECK(hipGraphInstantiate(&instance, graph, NULL, NULL, 0));
    
    CUDA_CHECK(hipEventRecord(stop, stream)); 
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&graphCreateTime, start, stop));
    
    for(int istep = 0; istep < NSTEP-1; istep++) {
        // Modifying buffers
        for (int i = 0; i < N; i++) {
            h_a1[i] += 1.0f; // or any other modification
            h_a2[i] += 1.0f;
            h_a3[i] += 1.0f;
            h_a4[i] += 1.0f;
            h_a5[i] += 1.0f;
            h_a6[i] += 1.0f;
            h_a7[i] += 1.0f;
            h_a8[i] += 1.0f;
            h_a9[i] += 1.0f;
            h_a10[i] += 1.0f;
        }

        // Start Timer
        CUDA_CHECK(hipEventRecord(start, stream));

        // Launch Graph
        CUDA_CHECK(hipGraphLaunch(instance, stream));
        // Synchronize the stream to ensure all operations are complete
        CUDA_CHECK(hipStreamSynchronize(stream));
        // for(int j = 0; j < N; j++) {
        //  printf("RESULT %d: %f\n", j, h_result[j]);
        // }
        // End Timer
        CUDA_CHECK(hipEventRecord(stop, stream));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
        if(istep >= skipBy){
            totalTime += elapsedTime;
            if(elapsedTime > upperTime) {
                upperTime = elapsedTime;
            }
            if(elapsedTime < lowerTime) {
                lowerTime = elapsedTime;
            }
            if(istep == skipBy){
                lowerTime = elapsedTime;
            }
        }
        //std::cout << "Elapsed time " << istep << ": " << elapsedTime << "ms" << std::endl;
    }
    // Time Calculations
    float AverageTime = (totalTime + graphCreateTime) / (NSTEP - skipBy);
    std::cout << "Average Time: " << AverageTime << "ms" << std::endl;
    std::cout << "Time Spread: " << upperTime <<  " - " << lowerTime << "ms" << std::endl;
    std::cout << "Total Time without Graph Creation: " << totalTime << "ms" << std::endl;
    std::cout << "Total Time with Graph Creation: " << (totalTime + graphCreateTime) << "ms" << std::endl;
    // Verify the result on the host
    int correct = 1;
    for (int i = 0; i < N; i++) {
        float expected = h_a1[i] + h_a2[i] + h_a3[i] + h_a4[i] + h_a5[i]
                       + h_a6[i] + h_a7[i] + h_a8[i] + h_a9[i] + h_a10[i];
        //printf("Test expected: %f\n", expected);
        //printf("Test result: %f\n", h_result[i]);
        if (h_result[i] != expected) {
            correct = 0;
            printf("Error at index %d: Expected %f, got %f\n", i, expected, h_result[i]);
            break;
        }
    }

    if (correct) {
        printf("Test PASSED\n");
    } else {
        printf("Test FAILED\n");
    }

    // Destroy the graph and exec object
    CUDA_CHECK(hipGraphDestroy(graph));
    CUDA_CHECK(hipGraphExecDestroy(instance));

    // Destroy the stream
    hipStreamDestroy(stream);

    // Free device memory
    hipFree(d_a1);
    hipFree(d_a2);
    hipFree(d_a3);
    hipFree(d_a4);
    hipFree(d_a5);
    hipFree(d_a6);
    hipFree(d_a7);
    hipFree(d_a8);
    hipFree(d_a9);
    hipFree(d_a10);
    hipFree(d_result);

    // Free host memory
    free(h_a1);
    free(h_a2);
    free(h_a3);
    free(h_a4);
    free(h_a5);
    free(h_a6);
    free(h_a7);
    free(h_a8);
    free(h_a9);
    free(h_a10);
    free(h_result);
    
    return 0;
}

